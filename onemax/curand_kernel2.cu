
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel(hiprandState *state, uint64_t seed)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(seed, tid, 0, &state[tid]);
}

__global__ void generate_randoms(hiprandState *globalState, float *randoms)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState localState = globalState[tid];
	randoms[tid * 2 + 0] = hiprand_uniform(&localState);
	randoms[tid * 2 + 1] = hiprand_uniform(&localState);
}

int main()
{
	int threads = 256;
	int blocks = 5120;
	int threadCount = blocks * threads;
	int N = blocks * threads * 2;

	hiprandState *dev_curand_states;
	float *dev_randomValues;
	float *host_randomValues;
	int *host_int;

	host_randomValues = (float *)malloc(N * sizeof(float));
	host_int = (int *)malloc(N * sizeof(float));

	hipMalloc(&dev_curand_states, threadCount * sizeof(hiprandState));
	hipMalloc(&dev_randomValues, N * sizeof(float));

	generate_randoms<<<blocks, threads>>>(dev_curand_states, dev_randomValues);

	hipMemcpy(host_randomValues, dev_randomValues, N * sizeof(float), hipMemcpyDeviceToHost);

	for (int i=0; i<N; ++i)
	{
		if (i < 8)
		{
			printf("%.8f, ", host_randomValues[i]);
		}
	}

	hipFree(dev_curand_states);
	hipFree(dev_randomValues);

	free(host_randomValues);
	free(host_int);

	return 0;
}

