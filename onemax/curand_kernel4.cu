
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__global__ void test_curand()
{
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState rgnState;
	hiprand_init(1234, tid, 0, &rgnState);

    float x = hiprand_uniform(&rgnState);
    float y = hiprand_uniform(&rgnState);
	printf("%f,%f\n", x, y);
}

int main(void)
{
	printf("start\n");
	test_curand<<<1, 1>>>();
	printf("end\n");
	return 0;
}
