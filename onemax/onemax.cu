#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <bitset>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#define POPSIZE 8
#define CHROMOSOME 16
// #define POPSIZE 512
// #define CHROMOSOME 512
#define NUM_OF_GENERATIONS 100
#define MUTATION_RATE 0.05
#define TOURNAMENT_SIZE 1
#define ELITISM true

#define N (POPSIZE * CHROMOSOME)
#define Nbytes (N*sizeof(int))
#define NT CHROMOSOME
#define NB POPSIZE
// #define NT (256)
// #define NB (N / NT) // 1より大きくなる

enum PARENTS {
	MALE   = 0,
	FEMALE = 1,
};

#define CUDA_CALL(x) do { if((x) != hipSuccess) {  \
	printf("Error at %s:%d\n", __FILE__, __LINE__); \
	return EXIT_FAILURE;}} while (0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) {  \
	printf("Error at %s:%d\n", __FILE__, __LINE__); \
	return EXIT_FAILURE;}} while (0)


__global__ void reduction(int *idata, int *odata)
{
    // スレッドと配列の要素の対応
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // スレッド番号
    int tx = threadIdx.x;
    int stride; // "隣"の配列要素まで距離

    // コンパイラの最適化を抑制
    // 複数のスレッドからアクセスされる変数に対する最適化
    // コンパイラが不要と判断して処理を削除してしまうことが有り、
    // 複数スレッドが変数の値をプライベートな領域にコピーして
    // 書き戻さない等が発生してしまう-->なのでvolatileを指定する
    // externを共有メモリの宣言に追加
    extern __shared__ volatile int s_idata[]; // 共有メモリの宣言

    s_idata[tx] = idata[i]; // グローバルメモリから共有メモリへデータをコピー
    __syncthreads(); // 共有メモリのデータは全スレッドから参照されるので同期を取る
    
    // ストライドを2倍し、ストライドがN/2以下ならループを継続
    // <<= : シフト演算の代入演算子 a <<= 1 --> a = a << 1と同じ
    // 最終stepではstrideが配列要素数のN/2となるので、strideがN/2
    // より大きくなるとループを中断
    for (stride = 1; stride <= blockDim.x/2; stride <<= 1)
    {
        // 処理を行うスレッドを選択
        if (tx % (2 * stride) == 0)
        {
            s_idata[tx] = s_idata[tx] + s_idata[tx + stride];
        }
        __syncthreads(); // スレッド間の同期を取る
        // stride = stride * 2; // ストライドを2倍して次のstepに備える
    }
    if (tx == 0) // 各ブロックのスレッド0が総和を出力用変数odataに書き込んで終了
    {
        odata[blockIdx.x] = s_idata[0];
    }
}

__host__ __device__ int getBest()
{
	return 0;
}

__global__ void setup_kernel(hiprandState *state)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState *state, float *result)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	float x;

	hiprandState localState = state[id];
	
	x = hiprand_uniform(&localState);

	state[id] = localState;
	result[id] = x;
}

__global__ void evaluation(int *population, int *fitness)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	int stride;

	extern __shared__ volatile int s_idata[];
	s_idata[tx] = population[i];
	__syncthreads();

	for (stride = 1; stride <= blockDim.x/2; stride <<= 1)
	{
		if (tx % (2 * stride) == 0)
		{
			s_idata[tx] = s_idata[tx] + s_idata[tx + stride];
		}
		__syncthreads();
	}

	if (tx == 0)
	{
		fitness[blockIdx.x] = s_idata[0];
	}
}

__host__ __device__ int getBestIndividual(const int *fitness)
{
	int best = 0;
	int best_index = 0;
	for (int i = 0; i < TOURNAMENT_SIZE; ++i)
	{
		if (fitness[i] > best)
		{
			best = fitness[i];
			best_index = i;
		}
	}
// #ifdef _DEBUG
// 	for (int i = 0; i < TOURNAMENT_SIZE; ++i)
// 	{
// 		printf("%d:%d ", i, fitness[i]);
// 	}
// 	printf("\n");
// #endif // _DEBUG

	return best_index;
}

__device__ int tournamentSelection(const int *fitness, hiprandState *dev_States, const int &id, PARENTS mf)
{
	int best_id;
	int tournament_individuals[TOURNAMENT_SIZE];
	int tournament_fitness[TOURNAMENT_SIZE];
	unsigned int random_id;
	unsigned int offset = (POPSIZE * TOURNAMENT_SIZE) * mf;

	for (int i = 0; i < TOURNAMENT_SIZE; ++i)
	{
		// hiprand_uniform returns random number uniformly distributed between (0, 1].
		// printf("--- id: %d ---\n", id * TOURNAMENT_SIZE + i +offset);
		hiprandState localState = dev_States[id + i + offset];
		// hiprandState localState = dev_States[id * TOURNAMENT_SIZE + i + offset];
		random_id = (unsigned int)(hiprand_uniform(&localState) * (POPSIZE-1));
		tournament_individuals[i] = random_id;
		tournament_fitness[i] = fitness[random_id];
	}
	best_id = getBestIndividual(tournament_fitness);

// #ifdef _DEBUG
// 	printf("%d,", id);
// 	for (int i=0; i<TOURNAMENT_SIZE; ++i)
// 	{
// 		printf("%d,%d,", tournament_individuals[i], tournament_fitness[i]);
// 	}
// 	printf("%d\n", tournament_individuals[best_id]);
// #endif // _DEBUG

	return tournament_individuals[best_id];
}


__global__ void selection(int* fitness, hiprandState *dev_States,
		int* parent1, int* parent2)
{
	int bx = blockIdx.x * blockDim.x;
	// int tx = threadIdx.x;
	// int id = blockIdx.x * blockDim.x + threadIdx.x;
	//if (id >= POPSIZE) return;

	parent1[bx] = tournamentSelection(fitness, dev_States, bx, MALE);
	parent2[bx] = tournamentSelection(fitness, dev_States, bx, FEMALE);
}

__global__ void crossover()
{
}

__global__ void mutation()
{
}

int my_rand(void)
{
    static thrust::default_random_engine rng;
    static thrust::uniform_int_distribution<int> dist(0, 1);

    return dist(rng);
}

void initializePopulationOnCPU(int *population)
{
    thrust::generate(population, population + N, my_rand);

#ifdef _DEBUG
    for (int i=0; i<POPSIZE; ++i)
	{
		for (int j=0; j<CHROMOSOME; ++j)
		{
			std::cout << population[i * POPSIZE + j];
		}
		std::cout << std::endl;
	}
#endif // _DEBUG
}

int main()
{
    //- GPU用変数 idata: 入力、odata: 出力(総和) --------------------------------------------------
    int *pdev_Population;
    int *pdev_Parent1;
    int *pdev_Parent2;
	thrust::device_vector<int> dev_Fitnesses(POPSIZE);
	thrust::device_vector<int> dev_Ranks(POPSIZE);

	int *pdev_Fitness = thrust::raw_pointer_cast(&dev_Fitnesses[0]);
	int *pdev_Ranks = thrust::raw_pointer_cast(&dev_Ranks[0]);
	thrust::sequence(dev_Ranks.begin(), dev_Ranks.end());

    hipMalloc((void **)&pdev_Population, Nbytes);
    hipMalloc((void **)&pdev_Parent1, NB * sizeof(int));
    hipMalloc((void **)&pdev_Parent2, NB * sizeof(int));

    //- CPU用変数 ---------------------------------------------------------------------------------
    int *phost_Population;
	int *phost_Fitness;
	int *phost_Ranks;
	int *phost_Parent1;
	int *phost_Parent2;

    phost_Fitness = (int *)malloc(POPSIZE * sizeof(int));
	phost_Ranks   = (int *)malloc(POPSIZE * sizeof(int));
	phost_Parent1 = (int *)malloc(POPSIZE * sizeof(int));
	phost_Parent2 = (int *)malloc(POPSIZE * sizeof(int));

	//- 乱数用変数 --------------------------------------------------------------------------------
	hiprandState *dev_States;
	hipMalloc((void **)&dev_States, POPSIZE * TOURNAMENT_SIZE * 2 * sizeof(hiprandState));
	// hipMalloc((void **)&dev_States, POPSIZE * sizeof(hiprandState));
	hipDeviceSynchronize();


	//- Preparation -------------------------------------------------------------------------------

    // CPU側でデータを初期化してGPUへコピー
    phost_Population = (int *)malloc(Nbytes);
    initializePopulationOnCPU(phost_Population);
    hipMemcpy(pdev_Population, phost_Population, Nbytes, hipMemcpyHostToDevice);

	// --------------------------------
	// Main loop
	// --------------------------------

	// initialize random numbers array for tournament selection
	// 乱数はトーナメントセレクションで用いられるので、個体の数だけあれば良い
	setup_kernel<<<POPSIZE, TOURNAMENT_SIZE * 2>>>(dev_States);
	hipDeviceSynchronize();

	evaluation<<<NB, NT, NT*sizeof(int)>>>(pdev_Population, pdev_Fitness);
	selection<<<POPSIZE, 1>>>(pdev_Fitness, dev_States, pdev_Parent1, pdev_Parent2);
	// selection<<<POPSIZE, TOURNAMENT_SIZE>>>(pdev_Fitness, dev_States, pdev_Parent1, pdev_Parent2);

	for (int i = 0; i < NUM_OF_GENERATIONS; ++i)
	{
		// selection<<<NB, NT, NT*sizeof(int)>>>();
		// crossover<<<NB, NT>>>();
		// mutation<<<NB, NT>>>();
		// evaluation<<<NB, NT, NT*sizeof(int)>>>(pdev_Population, pdev_Fitness);
	}

    hipMemcpy(phost_Fitness, pdev_Fitness, POPSIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(phost_Parent1, pdev_Parent1, POPSIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(phost_Parent2, pdev_Parent2, POPSIZE * sizeof(int), hipMemcpyDeviceToHost);

	// hipMemcpy(phost_Ranks, pdev_Ranks, POPSIZE * sizeof(int), hipMemcpyHostToHost);

#ifdef _DEBUG
	std::cout << "Fitness" << std::endl;
    for (int i=0; i < POPSIZE; ++i) std::cout << phost_Fitness[i] << ",";
	std::cout << std::endl << "Parent1" << std::endl;
	for (int i=0; i < POPSIZE; ++i) std::cout << phost_Parent1[i] << ",";
	std::cout << std::endl << "Parent2" << std::endl;
	for (int i=0; i < POPSIZE; ++i) std::cout << phost_Parent2[i] << ",";
	std::cout << std::endl;
#endif // _DEBUG

    // printf("sum = %d\n", sum);
    hipFree(pdev_Population);
    // hipFree(pdev_Fitness); thrust
	// hipFree(pdev_Ranks); thrust

    free(phost_Population);
	free(phost_Fitness);
	free(phost_Ranks);
	free(phost_Parent1);
	free(phost_Parent2);

    return 0;
}
