#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "Parameters.hpp"
#include "CUDAKernels.h"
#include "Misc.h"
int main()
{
    // 実行時間計測用
    float elapsed_time = 0.0f;
    // イベントを取り扱う変数
    hipEvent_t start, end;
    // イベントのクリエイト
    hipEventCreate(&start);
    hipEventCreate(&end);


    // パラメータ読み込み
    Parameters *prms = new Parameters();
    prms->loadParams();

    int host_popsize = prms->getPopsize();
    // printf("host_popsize %d\n", host_popsize);
    copyToDevice(prms->getEvoPrms());
    // dev_prms_show<<<1, 1>>>();
	// hipDeviceSynchronize();

    const int POPSIZE = prms->getPopsize();
    const int CHROMOSOME = prms->getChromosome();
    const int NUM_OF_GENERATIONS = prms->getNumOfGenerations();
    const int NUM_OF_ELITE = prms->getNumOfElite();
    const int TOURNAMENT_SIZE = prms->getTournamentSize();
    const int NUM_OF_CROSSOVER_POINTS  = prms->getNumOfCrossoverPoints();
    const float MUTATION_RATE = prms->getMutationRate();
    const int N = POPSIZE * CHROMOSOME;
    const int Nbytes = N * sizeof(int);

    //- GPU用変数 idata: 入力、odata: 出力(総和) --------------------------------------------------
	thrust::device_vector<int> dev_PopulationOdd(N);
	thrust::device_vector<int> dev_PopulationEven(N);
	thrust::device_vector<int> dev_Parent1(POPSIZE);
	thrust::device_vector<int> dev_Parent2(POPSIZE);
	thrust::device_vector<int> dev_Fitnesses(POPSIZE);
	thrust::device_vector<int> dev_SortedFitnesses(POPSIZE);
	thrust::device_vector<int> dev_SortedId(POPSIZE);
    thrust::device_vector<int> dev_TournamentIndividuals(TOURNAMENT_SIZE);
    thrust::device_vector<int> dev_TournamentFitness(TOURNAMENT_SIZE);

	int *pdev_PopulationOdd = thrust::raw_pointer_cast(&dev_PopulationOdd[0]);
	int *pdev_PopulationEven = thrust::raw_pointer_cast(&dev_PopulationEven[0]);
	int *pdev_Parent1 = thrust::raw_pointer_cast(&dev_Parent1[0]);
	int *pdev_Parent2 = thrust::raw_pointer_cast(&dev_Parent2[0]);
	int *pdev_Fitness = thrust::raw_pointer_cast(&dev_Fitnesses[0]);
	int *pdev_SortedFitness = thrust::raw_pointer_cast(&dev_SortedFitnesses[0]);
	int *pdev_SortedId = thrust::raw_pointer_cast(&dev_SortedId[0]);
    int *pdev_TournamentIndividuals = thrust::raw_pointer_cast(&dev_TournamentIndividuals[0]);
    int *pdev_TournamentFitness = thrust::raw_pointer_cast(&dev_TournamentFitness[0]);

    //- CPU用変数 ---------------------------------------------------------------------------------
    int *phost_Population;
	int *phost_Fitness;
	int *phost_SortedId;
	int *phost_Parent1;
	int *phost_Parent2;

    phost_Fitness       = (int *)malloc(POPSIZE * sizeof(int));
	phost_SortedId   = (int *)malloc(POPSIZE * sizeof(int));
	phost_Parent1       = (int *)malloc(POPSIZE * sizeof(int));
	phost_Parent2       = (int *)malloc(POPSIZE * sizeof(int));

	//- 乱数用変数 --------------------------------------------------------------------------------
	hiprandState *dev_TournamentStates;
	hipMalloc((void **)&dev_TournamentStates, POPSIZE * TOURNAMENT_SIZE * 2 * NUM_OF_GENERATIONS * sizeof(hiprandState));

	hiprandState *dev_CrossoverStates;
	hipMalloc((void **)&dev_CrossoverStates, POPSIZE * NUM_OF_CROSSOVER_POINTS * NUM_OF_GENERATIONS * sizeof(hiprandState));

	hiprandState *dev_MutationStates;
	hipMalloc((void **)&dev_MutationStates, POPSIZE * CHROMOSOME * NUM_OF_GENERATIONS * sizeof(hiprandState));

	//- Preparation -------------------------------------------------------------------------------

    // CPU側でデータを初期化してGPUへコピー
    phost_Population = (int *)malloc(POPSIZE * CHROMOSOME * sizeof(int));
    initializePopulationOnCPU(phost_Population, prms);
#ifdef _DEBUG
	for (int i = 0; i < POPSIZE; ++i)
	{
		for (int j = 0; j < CHROMOSOME; ++j)
		{
			printf("%d", phost_Population[i * CHROMOSOME + j]);
		}
		printf("\n");
	}
#endif // _DEBUG
    hipMemcpy(pdev_PopulationEven, phost_Population, Nbytes, hipMemcpyHostToDevice);

	// --------------------------------
	// Main loop
	// --------------------------------

    // 実行時間測定開始
    hipEventRecord(start, 0);

	// initialize random numbers array for tournament selection
	// 乱数はトーナメントセレクションで用いられるので、個体の数x2だけあれば良い
	setup_kernel<<<POPSIZE * NUM_OF_GENERATIONS, TOURNAMENT_SIZE * 2>>>(dev_TournamentStates);
	hipDeviceSynchronize();

	setup_kernel<<<POPSIZE * NUM_OF_GENERATIONS, NUM_OF_CROSSOVER_POINTS>>>(dev_CrossoverStates);
	hipDeviceSynchronize();

	setup_kernel<<<POPSIZE * NUM_OF_GENERATIONS, CHROMOSOME>>>(dev_MutationStates);
	hipDeviceSynchronize();

	evaluation<<<POPSIZE, CHROMOSOME, CHROMOSOME*sizeof(int)>>>(pdev_PopulationEven, pdev_Fitness);
	hipDeviceSynchronize();

	// dev_show<<<1, POPSIZE>>>(pdev_PopulationEven, pdev_Fitness, pdev_SortedFitness, pdev_Parent1, pdev_Parent2);
	// hipDeviceSynchronize();

	// mutation<<<POPSIZE, CHROMOSOME>>>(pdev_PopulationEven, dev_MutationStates, 0);

	for (int gen = 0; gen < NUM_OF_GENERATIONS; ++gen)
	{
		// printf("#####Gen: %d #######\n", gen);

		thrust::copy(thrust::device, dev_Fitnesses.begin(), dev_Fitnesses.end(), dev_SortedFitnesses.begin());
		thrust::sequence(dev_SortedId.begin(), dev_SortedId.end());
		thrust::sort_by_key(dev_SortedFitnesses.begin(), dev_SortedFitnesses.end(), dev_SortedId.begin()); 

		selection<<<1, POPSIZE>>>(
		// selection<<<N/POPSIZE, POPSIZE>>>(                                                          
				pdev_Fitness,
				pdev_SortedId,
				dev_TournamentStates,
				pdev_Parent1,
				pdev_Parent2,
				gen,
                pdev_TournamentIndividuals,
                pdev_TournamentFitness);
		hipDeviceSynchronize();

		if (gen % 2 == 0) // Even
		{
			crossover<<<1, POPSIZE, POPSIZE * sizeof(int) * 2>>>(
			// crossover<<<N/POPSIZE, POPSIZE, POPSIZE * sizeof(int) * 2>>>(
					pdev_PopulationEven,
					pdev_PopulationOdd,
					dev_CrossoverStates,
					pdev_Parent1,
					pdev_Parent2,
					gen);
			hipDeviceSynchronize();

			mutation<<<POPSIZE, CHROMOSOME>>>(pdev_PopulationOdd, dev_MutationStates, gen);
			hipDeviceSynchronize();

			evaluation<<<POPSIZE, CHROMOSOME, CHROMOSOME*sizeof(int)>>>(pdev_PopulationOdd, pdev_Fitness);
			hipDeviceSynchronize();
		}
		else // Odd
		{
			crossover<<<1, POPSIZE, POPSIZE * sizeof(int) * 2>>>(
			// crossover<<<N/POPSIZE, POPSIZE, POPSIZE * sizeof(int) * 2>>>(
					pdev_PopulationOdd,
					pdev_PopulationEven,
					dev_CrossoverStates,
					pdev_Parent1,
					pdev_Parent2,
					gen);
			hipDeviceSynchronize();

			mutation<<<POPSIZE, CHROMOSOME>>>(pdev_PopulationEven, dev_MutationStates, gen);
			hipDeviceSynchronize();

			evaluation<<<POPSIZE, CHROMOSOME, CHROMOSOME*sizeof(int)>>>(pdev_PopulationEven, pdev_Fitness);
			hipDeviceSynchronize();
		}
#ifdef _TREND
        hipMemcpy(phost_Fitness,  pdev_Fitness,  POPSIZE * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(phost_SortedId, pdev_SortedId, POPSIZE * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(phost_Parent1,  pdev_Parent1,  POPSIZE * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(phost_Parent2,  pdev_Parent2,  POPSIZE * sizeof(int), hipMemcpyDeviceToHost);
        if (gen % 2 == 0)
        {
            hipMemcpy(phost_Population, pdev_PopulationOdd, Nbytes, hipMemcpyDeviceToHost);
        }
        else
        {
            hipMemcpy(phost_Population, pdev_PopulationEven, Nbytes, hipMemcpyDeviceToHost);
        }
        // showPopulationOnCPU(phost_Population, phost_Fitness, phost_Parent1, phost_Parent2, prms);
        showSummaryOnCPU(gen, phost_Fitness, prms);
#endif // _TREND
	}

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);
    // std::cout << "Elapsed Time: " << elapsed_time << std::endl;
    std::cout << POPSIZE << "," << CHROMOSOME << "," << elapsed_time << std::endl;

    hipMemcpy(phost_Fitness, pdev_Fitness, POPSIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(phost_Parent1, pdev_Parent1, POPSIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(phost_Parent2, pdev_Parent2, POPSIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(phost_Population, pdev_PopulationOdd, Nbytes, hipMemcpyDeviceToHost);

    free(phost_Population);
	free(phost_Fitness);
	free(phost_SortedId);
	free(phost_Parent1);
	free(phost_Parent2);
    delete prms;

    return 0;
}
