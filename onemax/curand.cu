
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand.h>

#define N 1024

/**
 * CPUから関数を呼ぶ場合
 */
int main()
{
	int i;
	float *value, *value_d;

	// generatorを宣言
	hiprandGenerator_t gen;
	hipMalloc((void**)&value_d, N * sizeof(float));
	// Generatorの生成
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	// 一様乱数を生成
	hiprandGenerateUniform(gen, value_d, N);
	value = (float *)malloc(N * sizeof(float));
	hipMemcpy(value, value_d, N * sizeof(float), hipMemcpyDeviceToHost);

	for (i=0; i<N; i++)
	{
		printf("%f\n", value[i]);
	}

	hiprandDestroyGenerator(gen);
	hipFree(value_d);
	free(value);
	return 0;
}
