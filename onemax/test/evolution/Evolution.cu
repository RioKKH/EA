#include <stdio.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Evolution.h"
#include "CUDAKernels.h"
#include "Parameters.h"

/**
 * Constructor of the class
 */
GPUEvolution::GPUEvolution()
    : mRandomSeed(0), mDeviceIdx(0)
{
}

GPUEvolution::GPUEvolution(Parameters* prms)
    : mRandomSeed(0),
      mDeviceIdx(0)
{
    printf("constructor\n");
    //- Select device
    // hipSetDevice(mDeviceIdx);
    // checkAndReportCudaError(__FILE__, __LINE__);

    //- Get parameters of the device
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, mDeviceIdx);

    // Create populations on CPU
    mHostParentPopulation    = new CPUPopulation(prms->getPopsize(), prms->getChromosome(), prms->getNumOfElite());
    mHostOffspringPopulation = new CPUPopulation(prms->getPopsize(), prms->getChromosome(), prms->getNumOfElite());

#ifdef _DEBUG
    for (int i = 0; i < prms->getPopsize(); ++i)
    {
        printf("%d,", i);
        for (int j = 0; j < prms->getChromosome(); ++j)
        {
            printf("%d", mHostParentPopulation->getDeviceData()->population[i * prms->getChromosome() + j]);
        }
        printf(":%d\n", mHostParentPopulation->getDeviceData()->fitness[i]);
    }
#endif // _DEBUG

    // Create populations on GPU
    mDevParentPopulation    = new GPUPopulation(prms->getPopsize(), prms->getChromosome(), prms->getNumOfElite());
    mDevOffspringPopulation = new GPUPopulation(prms->getPopsize(), prms->getChromosome(), prms->getNumOfElite());

    // Copy population from CPU to GPU
    mDevParentPopulation->copyToDevice(mHostParentPopulation->getDeviceData());
    mDevOffspringPopulation->copyToDevice(mHostOffspringPopulation->getDeviceData());

    mMultiprocessorCount = prop.multiProcessorCount;
    // mParams.setNumberOfDeviceSMs(prop.multiProcessorCount);

    // Create statistics
    // mStatistics = new GPUStatistics();

    // Initialize Random seed
    initRandomSeed();
    printf("end of constructor\n");
}; // end of GPUEvolution


/**
 * Destructor of the class
 */
GPUEvolution::~GPUEvolution()
{
    delete mHostParentPopulation;
    delete mHostOffspringPopulation;
    delete mDevParentPopulation;
    delete mDevOffspringPopulation;
} // end of Destructor


/**
 * Run Evolution
 */
void GPUEvolution::run(Parameters* prms)
{
    std::uint16_t generation = 0;
    initialize(prms);

    showPopulation(prms, generation);

    for (generation = 0; generation < prms->getNumOfGenerations(); ++generation)
    {
        printf("### Number of Generations : %d ###\n", generation);
        runEvolutionCycle(prms);
        showPopulation(prms, generation);
    }
}


void GPUEvolution::initRandomSeed()
{
    struct timeval tp1;
    gettimeofday(&tp1, nullptr);
    mRandomSeed = (tp1.tv_sec / (mDeviceIdx + 1)) * tp1.tv_usec;
#ifdef _DEBUG
    printf("mRandomSeed: %d\n", mRandomSeed);
#endif // _DEBUG
}

/**
 * Initialization of the GA
 */
void GPUEvolution::initialize(Parameters* prms)
{
    copyToDevice(prms->getEvoPrms());

    dim3 blocks;
    dim3 threads;


    //- 初期集団生成 ------------------------------------------------------------------------------
    blocks.x  = prms->getPopsize() / 2;
    blocks.y  = 1; blocks.z  = 1;

    threads.x = prms->getChromosome();
    threads.y = 1;
    threads.z = 1;

    cudaGenerateFirstPopulationKernel<<<blocks, threads>>>
                                     (mDevParentPopulation->getDeviceData(), getRandomSeed());
    checkAndReportCudaError(__FILE__, __LINE__);


    //- Fitness評価 -------------------------------------------------------------------------------
    blocks.x  = prms->getPopsize();
    blocks.y  = 1;
    blocks.z  = 1;

    threads.x = prms->getChromosome();
    threads.y = 1;
    threads.z = 1;

    evaluation<<<blocks, threads, prms->getChromosome() * sizeof(int)>>>(mDevParentPopulation->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);

    evaluation<<<blocks, threads, prms->getChromosome() * sizeof(int)>>>(mDevOffspringPopulation->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);


    //- 疑似エリート保存戦略 ----------------------------------------------------------------------
    blocks.x  = prms->getNumOfElite() * 2;
    blocks.y  = 1;
    blocks.z  = 1;

    threads.x = prms->getPopsize() / prms->getNumOfElite();
    threads.y = 1;
    threads.z = 1;

#ifdef _DEBUG
    printf("blocks.x:%d, threads.x:%d, offset:%d, shared_memory_size:%d\n",
            blocks.x, threads.x, blocks.x * threads.x / 2, prms->getPopsize() * 2);
#endif // _DEBUG

    pseudo_elitism<<<blocks, threads, threads.x * 2 * sizeof(int)>>>(mDevParentPopulation->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);

    pseudo_elitism<<<blocks, threads, threads.x * 2 * sizeof(int)>>>(mDevOffspringPopulation->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);


} // end of initialize


/**
 * Run evolutionary cycle for defined number of generations
 */
void GPUEvolution::runEvolutionCycle(Parameters* prms)
{
    dim3 blocks;
    dim3 threads;

    //- Selection, Crossover, and Mutation ---------------------------------------------------------
    int CHR_PER_BLOCK = (prms->getPopsize() % WARP_SIZE == 0)
                         ? prms->getPopsize() / WARP_SIZE
                         : prms->getPopsize() / WARP_SIZE + 1;

    blocks.x = CHR_PER_BLOCK;
    blocks.y = 1;
    blocks.z = 1;

    threads.x = (prms->getPopsize() > WARP_SIZE) ? WARP_SIZE : prms->getPopsize();
    // threads.x = (prms->getPopsize() > WARP_SIZE) ? WARP_SIZE : prms->getPopsize() / 2;
    threads.y = 1;
    threads.z = 1;

    int shared_memory_size =   prms->getPopsize()        * sizeof(int)
                             + prms->getPopsize()        * sizeof(int)
                             + prms->getTournamentSize() * sizeof(int);

    printf("Start of cudaGeneticManipulationKernel\n");
    printf("GA: blocks: %d, threads: %d\n", blocks.x, threads.x);
    cudaGeneticManipulationKernel<<<blocks, threads, shared_memory_size>>> 
                                 (mDevParentPopulation->getDeviceData(),
                                  mDevOffspringPopulation->getDeviceData(),
                                  getRandomSeed());
    checkAndReportCudaError(__FILE__, __LINE__);
    printf("End of cudaGeneticManipulationKernel\n");


    //- Fitness評価 --------------------------------------------------------------------------------
    blocks.x  = prms->getPopsize();
    blocks.y  = 1;
    blocks.z  = 1;

    threads.x = prms->getChromosome();
    threads.y = 1;
    threads.z = 1;

    printf("Evaluation: blocks: %d, threads: %d\n", blocks.x, threads.x);
    evaluation<<<blocks, threads, prms->getChromosome() * sizeof(int)>>>(mDevParentPopulation->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);

    evaluation<<<blocks, threads, prms->getChromosome() * sizeof(int)>>>(mDevOffspringPopulation->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);


    //- 疑似エリート保存戦略 -----------------------------------------------------------------------
    blocks.x  = prms->getNumOfElite() * 2;
    blocks.y  = 1;
    blocks.z  = 1;

    threads.x = prms->getPopsize() / prms->getNumOfElite();
    threads.y = 1;
    threads.z = 1;
    printf("blocks.x:%d, threads.x:%d, offset:%d, shared_memory_size:%d\n",
            blocks.x, threads.x, blocks.x * threads.x / 2, prms->getPopsize() * 2);

    pseudo_elitism<<<blocks, threads, threads.x * 2 * sizeof(int)>>>(mDevParentPopulation->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);

    pseudo_elitism<<<blocks, threads, threads.x * 2 * sizeof(int)>>>(mDevOffspringPopulation->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);


    //- 親と子の入れ替え & Elitesの差し込み --------------------------------------------------------
    printf("Copy population from offspring to parent, then insert elites in it.\n");
    blocks.x = 1; // gridDim.x
    // blocks.x = CHR_PER_BLOCK; // gridDim.x
    blocks.y = 1;
    blocks.z = 1;

    threads.x = 1; // blockDim.x
    // threads.x = prms->getPopsize() / CHR_PER_BLOCK; // blockDim.x
    threads.y = 1;
    threads.z = 1;
    swapPopulation<<<blocks, threads>>>(mDevParentPopulation->getDeviceData(),
                                        mDevOffspringPopulation->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);
}


void GPUEvolution::showPopulation(Parameters* prms, std::uint16_t generation)
{
    int csize = prms->getChromosome();
    int psize = prms->getPopsize();
    int esize = prms->getNumOfElite();

    mDevParentPopulation->copyFromDevice(mHostParentPopulation->getDeviceData());
    mDevOffspringPopulation->copyFromDevice(mHostOffspringPopulation->getDeviceData());

    printf("------------ Parent:%d ------------ \n", generation);
    // for (int k = 0; k < psize; ++k)
    for (int k = 0; k < esize; ++k)
    {
        printf("elite%d : %d\n", k, mHostParentPopulation->getDeviceData()->elitesIdx[k]);
    }
    printf("\n");

    for (int i = 0; i < psize; ++i)
    {
        printf("%d,", i);
        for (int j = 0; j < csize; ++j)
        {
            printf("%d", mHostParentPopulation->getDeviceData()->population[i * csize + j]);
        }
        printf(":%d\n", mHostParentPopulation->getDeviceData()->fitness[i]);
    }

    printf("------------ Offspring:%d ------------ \n", generation);
    for (int k = 0; k < esize; ++k)
    {
        printf("elite%d : %d\n", k, mHostOffspringPopulation->getDeviceData()->elitesIdx[k]);
    }
    printf("\n");
    
    for (int i = 0; i < psize; ++i)
    {
        printf("%d,", i);
        for (int j = 0; j < csize; ++j)
        {
            printf("%d", mHostOffspringPopulation->getDeviceData()->population[i * csize + j]);
        }
        printf(":%d\n", mHostOffspringPopulation->getDeviceData()->fitness[i]);
    }
}


