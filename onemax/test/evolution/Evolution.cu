#include <stdio.h>
#include <sys/time.h>

#include "Evolution.h"
// #include "CUDAKernels.h"
// #include "Parameters.h"

/**
 * Constructor of the class
 */
GPUEvolution::GPUEvolution()
    : mRandomSeed(0),
      mDeviceIdx(0)
{
    // Select device
    // hipSetDevice(mDeviceIdx);
    // checkAndReportCudaError(__FILE__, __LINE__);

    // Get parameters of the device
    hipDeviceProp_t prop;

    // hipGetDeviceProperties(&prop, mDeviceIdx);
    // checkAndReportCudaError(__FILE__, __LINE__);

    // multiprocessorCount = prop.multiProcessorCount;
    // mParams.setNumberOfDeviceSMs(prop.multiProcessorCount);

    // Load knapsack data from the file.
    // mGlobalData.LoadFromFile();

    // Create populations on GPU
    // mMasterPopulation = new GPUPopulation(mParams.getPopulationsSize(), mParams.getChromosomeSize());
    // mOffspringPopulation = new GPUPopulation(mParams.getOffspringPopulationsSize(), mParams.getChromosomeSize());

    // Create statistics
    // mStatistics = new GPUStatistics();

    // Initialize Random seed
    initRandomSeed();
}; // end of GPUEvolution


/**
 * Destructor of the class
 */
GPUEvolution::~GPUEvolution()
{
} // end of Destructor


/**
 * Run Evolution
 */
void GPUEvolution::run()
{
    // initialize();
    runEvolutionCycle();
}


void GPUEvolution::initRandomSeed()
{
    struct timeval tp1;
    gettimeofday(&tp1, nullptr);
    mRandomSeed = (tp1.tv_sec / (mDeviceIdx + 1)) * tp1.tv_usec;
}

/**
 * Initialization of the GA
 */
void GPUEvolution::initialize()
{
    /// TODO
} // end of initialize

/**
 * Run evolutionary cycle for defined number of generations
 */
void GPUEvolution::runEvolutionCycle()
{
    dim3 blocks;
    dim3 threads;

    // Every chromosome is treated by a single warp, theare are as many warps as individuals per block
    // threads.x = WARP_SIZE;
    // threads.y = CHR_PER_BLOCK;
    // threads.z = 1;

    for (int i = 0; i < 10; ++i)
    {
        printf("%d,%d\n", i, getRandomSeed());
    }
}

