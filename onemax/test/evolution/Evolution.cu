#include <stdio.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Evolution.h"
#include "CUDAKernels.h"
#include "Parameters.h"

/**
 * Constructor of the class
 */
GPUEvolution::GPUEvolution()
    : mRandomSeed(0), mDeviceIdx(0)
{
}

GPUEvolution::GPUEvolution(Parameters* prms)
    : mRandomSeed(0),
      mDeviceIdx(0)
{
    //- Select device
    // hipSetDevice(mDeviceIdx);
    // checkAndReportCudaError(__FILE__, __LINE__);

    //- Get parameters of the device
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, mDeviceIdx);

    // Create populations on CPU
    mHostPopulationOdd  = new CPUPopulation(prms->getPopsize(), prms->getChromosome(), prms->getNumOfElite());
    mHostPopulationEven = new CPUPopulation(prms->getPopsize(), prms->getChromosome(), prms->getNumOfElite());

    printf("check: %d, %d, %d\n",
            mHostPopulationOdd->getDeviceData()->chromosomeSize,
            mHostPopulationOdd->getDeviceData()->populationSize,
            mHostPopulationOdd->getDeviceData()->elitesSize);

    // Create populations on GPU
    mDevPopulationOdd  = new GPUPopulation(prms->getPopsize(), prms->getChromosome(), prms->getNumOfElite());
    mDevPopulationEven = new GPUPopulation(prms->getPopsize(), prms->getChromosome(), prms->getNumOfElite());

    // Copy population from CPU to GPU
    mDevPopulationEven->copyToDevice(mHostPopulationEven->getDeviceData());
    mDevPopulationOdd->copyToDevice(mHostPopulationOdd->getDeviceData());

    mMultiprocessorCount = prop.multiProcessorCount;
    // mParams.setNumberOfDeviceSMs(prop.multiProcessorCount);

    // Create statistics
    // mStatistics = new GPUStatistics();

    // Initialize Random seed
    initRandomSeed();
}; // end of GPUEvolution


/**
 * Destructor of the class
 */
GPUEvolution::~GPUEvolution()
{
    delete mHostPopulationEven;
    delete mHostPopulationOdd;
    delete mDevPopulationEven;
    delete mDevPopulationOdd;
} // end of Destructor


/**
 * Run Evolution
 */
void GPUEvolution::run(Parameters* prms)
{
    initialize(prms);

#ifdef _DEBUG
    showPopulation(prms);
#endif // _DEBUG

    // runEvolutionCycle(prms);
}


void GPUEvolution::initRandomSeed()
{
    struct timeval tp1;
    gettimeofday(&tp1, nullptr);
    mRandomSeed = (tp1.tv_sec / (mDeviceIdx + 1)) * tp1.tv_usec;
}

/**
 * Initialization of the GA
 */
void GPUEvolution::initialize(Parameters* prms)
{
    dim3 blocks;
    dim3 threads;

    copyToDevice(prms->getEvoPrms());

    cudaGenerateFirstPopulationKernel<<<mMultiprocessorCount * 2, 256>>>
                                     (mDevPopulationEven->getDeviceData(),
                                      getRandomSeed());
    checkAndReportCudaError(__FILE__, __LINE__);

    cudaGenerateFirstPopulationKernel<<<mMultiprocessorCount * 2, 256>>>
                                     (mDevPopulationOdd->getDeviceData(),
                                      getRandomSeed());
    checkAndReportCudaError(__FILE__, __LINE__);

    evaluation<<<prms->getPopsize(),
                 prms->getChromosome(),
                 prms->getChromosome() * sizeof(int)>>>(mDevPopulationEven->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);

    evaluation<<<prms->getPopsize(),
                 prms->getChromosome(),
                 prms->getChromosome() * sizeof(int)>>>(mDevPopulationOdd->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);

    // blocks.x  = prms->getNumOfElite();                      blocks.y  = 1; blocks.z  = 1;
    blocks.x  = prms->getNumOfElite() * 2;                  blocks.y  = 1; blocks.z  = 1;
    threads.x = prms->getPopsize() / prms->getNumOfElite(); threads.y = 1; threads.z = 1;
    printf("t:b = %d:%d\n", threads.x, blocks.x);

    pseudo_elitism<<<blocks, threads, prms->getPopsize() * 2 * sizeof(int)>>>(mDevPopulationEven->getDeviceData());
    // pseudo_elitism<<<blocks, threads, prms->getPopsize() * sizeof(int)>>>(mDevPopulationEven->getDeviceData());
    checkAndReportCudaError(__FILE__, __LINE__);

    // pseudo_elitism<<<blocks, threads, prms->getPopsize() * sizeof(int)>>>(mDevPopulationOdd->getDeviceData());
    // checkAndReportCudaError(__FILE__, __LINE__);

    // ホストとデバイスの同期を取る
    // hipDeviceSynchronize();

} // end of initialize

/**
 * Run evolutionary cycle for defined number of generations
 */
void GPUEvolution::runEvolutionCycle(Parameters* prms)
{
    dim3 blocks;
    dim3 threads;

    blocks.x = prms->getPopsize();
    blocks.y = 1;
    blocks.z = 1;

    // threads.x = WARP_SIZE;
    //threads.y = prms->getChromosome() / WARP_SIZE;
    threads.x = prms->getChromosome();
    threads.y = 1;
    threads.z = 1;

    const int POPSIZE = prms->getPopsize();
    printf("POPSIZE %d\n", POPSIZE);

    // Every chromosome is treated by a single warp, theare are as many warps as individuals per block
    // threads.x = WARP_SIZE;
    // threads.y = CHR_PER_BLOCK;
    // threads.z = 1;

    printf("Before cuda kernel\n");
    cudaCallRandomNumber<<<32, 4>>>(getRandomSeed());
    // cudaCallRandomNumber<<<blocks, threads>>>(getRandomSeed());
    hipDeviceSynchronize();
}

void GPUEvolution::showPopulation(Parameters* prms)
{
    int csize = prms->getChromosome();
    int psize = prms->getPopsize();
    int esize = prms->getNumOfElite();

    mDevPopulationEven->copyFromDevice(mHostPopulationEven->getDeviceData());
    mDevPopulationOdd->copyFromDevice(mHostPopulationOdd->getDeviceData());

    for (int k = 0; k < esize; ++k)
    {
        printf("%d,", mHostPopulationEven->getDeviceData()->elitesIdx[k]);
    }
    printf("\n");

    for (int i = 0; i < psize; ++i)
    {
        printf("%d,", i);
        for (int j = 0; j < csize; ++j)
        {
            printf("%d", mHostPopulationEven->getDeviceData()->population[psize * i + j]);
        }
        printf(":%d\n", mHostPopulationEven->getDeviceData()->fitness[i]);
        // for (int k = 0; k < esize; ++k)
        // {
        //     if (i == mHostPopulationEven->getDeviceData()->elitesIdx[k])
        //     {
        //         printf(",*\n");
        //     }
        //     else
        //     {
        //         printf("\n");
        //     }
        // }
    }
}


