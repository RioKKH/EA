#include "hip/hip_runtime.h"
#include "Random123/philox.h"

#include "CUDAKernels.h"

typedef r123::Philox2x32 RNG_2x32;
typedef r123::Philox4x32 RNG_4x32;

__device__ RNG_2x32::ctr_type generateTwoRndValues(unsigned int key,
                                                   unsigned int counter);

/*
inline __device__ int getIndex(unsigned int chromosomeIdx,
                               unsigned int geneIdx);
*/

inline __device__ RNG_2x32::ctr_type generateTwoRndValues(unsigned int key,
                                                               unsigned int counter)
{
    RNG_2x32 rng;

    return rng({0, counter}, {key});
} // end of TwoRandomINTs

/*
inline __device__ int getIndex(unsigned int chromosomeIdx,
                               unsigned int geneIdx)
{
    return (chromosomeIdx * gpuEvolutionParameters.chromosomeSize + geneIdx);
}
*/

__global__ void cudaCallRandomNumber(unsigned int randomSeed)
{
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
}

