#include <iostream>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "population.hpp"

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int main(int argc, char **argv)
{
    int gen_max = 0;
    int pop_size = 0;
    int chromosome = 0;

#ifdef _ELAPSED_TIME
    // 実行時間計測用
    float elapsed_time = 0.0f;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
#endif // _ELAPSED_TIME

    Parameters *prms;
    prms = new Parameters();
    gen_max = prms->getGenMax();
    pop_size = prms->getPopSize();
    chromosome = prms->getNumberOfChromosome();

    srand((unsigned int)time(NULL));

    population *pop;
    pop = new population(prms);

#ifdef _ELAPSED_TIME
    hipEventRecord(start, 0);
#endif // _ELAPSED_TIME
    for (int i = 1; i <= gen_max; i++) {
        pop->alternate();
    }
#ifdef _ELAPSED_TIME
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);
    std::cout << pop_size << "," << chromosome << "," << elapsed_time << std::endl;
#endif // _ELAPSED_TIME

    // delete pointers
    delete pop;
    delete prms;

    return 0;
}
